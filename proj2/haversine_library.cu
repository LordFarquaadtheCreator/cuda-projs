#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void haversine_distance_kernel(int size, const double *x1,const double *y1,
    const double *x2,const double *y2, double *dist)
{
  double deltaLat = floor(*x1 - *x2);
  double deltaLong = floor(*y1 - *y2);
  double cosinesMultiplied = cos(*x1) * cos(*x2);
  double sinDeltaLat = 2 * sin(deltaLat / 2) * cos(deltaLat / 2);
  double sinDeltaLong = 2 * sin(deltaLong / 2) * cos(deltaLong / 2);

  *dist = sinDeltaLat + cosinesMultiplied * sinDeltaLong;
  return;
}


void run_kernel(int size, const double *x1,const double *y1, const double *x2,const double *y2, double *dist)
   
{
  dim3 dimBlock(1024);
  printf("in run_kernel dimBlock.x=%d\n",dimBlock.x);

  dim3 dimGrid(ceil((double)size / dimBlock.x));
  
  haversine_distance_kernel<<<dimGrid, dimBlock>>>
    (size,x1,y1,x2,y2,dist);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::stringstream strstr;
    strstr << "run_kernel launch failed" << std::endl;
    strstr << "dimBlock: " << dimBlock.x << ", " << dimBlock.y << std::endl;
    strstr << "dimGrid: " << dimGrid.x << ", " << dimGrid.y << std::endl;
    strstr << hipGetErrorString(error);
    throw strstr.str();
  }
}
